#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "LaunchParams.h"

extern "C" __constant__ LaunchParams launchParams;

__global__ void __raygen__renderFrame()
{
	printf("doing ray generation in OptiX!");

	// TODO: write to framebuffer here
	launchParams.FramebufferData[0] = 0;
}

// dummy functions for OptiX pipeline
__global__ void __miss__radiance() {}
__global__ void __closesthit_radiance() {}